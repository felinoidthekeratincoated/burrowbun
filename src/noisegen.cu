#include "hip/hip_runtime.h"
// Modified from libnoise

// noisegen.cpp
//
// Copyright (C) 2003, 2004 Jason Bevins
//
// This library is free software; you can redistribute it and/or modify it
// under the terms of the GNU Lesser General Public License as published by
// the Free Software Foundation; either version 2.1 of the License, or (at
// your option) any later version.
//
// This library is distributed in the hope that it will be useful, but WITHOUT
// ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
// FITNESS FOR A PARTICULAR PURPOSE.  See the GNU Lesser General Public
// License (COPYING.txt) for more details.
//
// You should have received a copy of the GNU Lesser General Public License
// along with this library; if not, write to the Free Software Foundation,
// Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA
//
// The developer's email is jlbezigvins@gmzigail.com (for great email, take
// off every 'zig'.)
//
#include <hip/hip_runtime.h>
#include "noisegen.cuh"
#include "interp.cuh"
#include "vectortable.cuh"

// Constants used by the current version of libnoise.
__constant__ int X_NOISE_GEN = 1619;
__constant__ int Y_NOISE_GEN = 31337;
__constant__ const int Z_NOISE_GEN = 6971;
__constant__ const int SEED_NOISE_GEN = 1013;
__constant__ const int SHIFT_NOISE_GEN = 8;

// The default values from libnoise's perlin module
__constant__ int octaveCount = 6;
__constant__ double frequency = 1.0;
__constant__ double lacunarity = 2.0;
__constant__ double persistence = 0.5;

// For ridged multi
__constant__ double spectralWeights[6] = {1.0, 0.5, 0.25, 0.125, 0.0625, 0.03125};

// Slightly modified from libnoise's perlin.cpp
__device__
double cuda_noise::GetPerlinValue (double x, double y, double z, int seed)
{
  double value = 0.0;
  double signal = 0.0;
  double curPersistence = 1.0;
  double nx, ny, nz;
  int curSeed;

  x *= frequency;
  y *= frequency;
  z *= frequency;

  for (int curOctave = 0; curOctave < octaveCount; curOctave++) {

    // Make sure that these floating-point values have the same range as a 32-
    // bit integer so that we can pass them to the coherent-noise functions.
    nx = MakeInt32Range (x);
    ny = MakeInt32Range (y);
    nz = MakeInt32Range (z);

    // Get the coherent-noise value from the input value and add it to the
    // final result.
    curSeed = (seed + curOctave) & 0xffffffff;
    signal = GradientCoherentNoise3D (nx, ny, nz, curSeed);
    value += signal * curPersistence;

    // Prepare the next octave.
    x *= lacunarity;
    y *= lacunarity;
    z *= lacunarity;
    curPersistence *= persistence;
  }

  return value;
}

//Slightly modified from libnoise's RidgeMulti module
__device__
double cuda_noise::GetRidgedMultiValue(double x, double y, double z, int seed)
{
  x *= frequency;
  y *= frequency;
  z *= frequency;

  double signal = 0.0;
  double value  = 0.0;
  double weight = 1.0;

  // These parameters should be user-defined; they may be exposed in a
  // future version of libnoise.
  double offset = 1.0;
  double gain = 2.0;

  for (int curOctave = 0; curOctave < octaveCount; curOctave++) {

    // Make sure that these floating-point values have the same range as a 32-
    // bit integer so that we can pass them to the coherent-noise functions.
    double nx, ny, nz;
    nx = MakeInt32Range (x);
    ny = MakeInt32Range (y);
    nz = MakeInt32Range (z);

    // Get the coherent-noise value.
    int curSeed = (seed + curOctave) & 0x7fffffff;
    signal = GradientCoherentNoise3D (nx, ny, nz, curSeed);

    // Make the ridges.
    signal = fabs (signal);
    signal = offset - signal;

    // Square the signal to increase the sharpness of the ridges.
    signal *= signal;

    // The weighting from the previous octave is applied to the signal.
    // Larger values have higher weights, producing sharp points along the
    // ridges.
    signal *= weight;

    // Weight successive contributions by the previous signal.
    weight = signal * gain;
    if (weight > 1.0) {
      weight = 1.0;
    }
    if (weight < 0.0) {
      weight = 0.0;
    }

    // Add the signal to the output value.
    value += (signal * spectralWeights[curOctave]);

    // Go to the next octave.
    x *= lacunarity;
    y *= lacunarity;
    z *= lacunarity;
  }

  return (value * 1.25) - 1.0;
}

__device__
double cuda_noise::GradientCoherentNoise3D (double x, double y, double z, int seed)
{
  // Create a unit-length cube aligned along an integer boundary.  This cube
  // surrounds the input point.
  int x0 = (x > 0.0? (int)x: (int)x - 1);
  int x1 = x0 + 1;
  int y0 = (y > 0.0? (int)y: (int)y - 1);
  int y1 = y0 + 1;
  int z0 = (z > 0.0? (int)z: (int)z - 1);
  int z1 = z0 + 1;

  // Map the difference between the coordinates of the input value and the
  // coordinates of the cube's outer-lower-left vertex onto an S-curve.
  double xs = 0, ys = 0, zs = 0;

  xs = SCurve3 (x - (double)x0);
  ys = SCurve3 (y - (double)y0);
  zs = SCurve3 (z - (double)z0);

  // Now calculate the noise values at each vertex of the cube.  To generate
  // the coherent-noise value at the input point, interpolate these eight
  // noise values using the S-curve value as the interpolant (trilinear
  // interpolation.)
  double n0, n1, ix0, ix1, iy0, iy1;
  n0   = GradientNoise3D (x, y, z, x0, y0, z0, seed);
  n1   = GradientNoise3D (x, y, z, x1, y0, z0, seed);
  ix0  = LinearInterp (n0, n1, xs);
  n0   = GradientNoise3D (x, y, z, x0, y1, z0, seed);
  n1   = GradientNoise3D (x, y, z, x1, y1, z0, seed);
  ix1  = LinearInterp (n0, n1, xs);
  iy0  = LinearInterp (ix0, ix1, ys);
  n0   = GradientNoise3D (x, y, z, x0, y0, z1, seed);
  n1   = GradientNoise3D (x, y, z, x1, y0, z1, seed);
  ix0  = LinearInterp (n0, n1, xs);
  n0   = GradientNoise3D (x, y, z, x0, y1, z1, seed);
  n1   = GradientNoise3D (x, y, z, x1, y1, z1, seed);
  ix1  = LinearInterp (n0, n1, xs);
  iy1  = LinearInterp (ix0, ix1, ys);

  return LinearInterp (iy0, iy1, zs);
}

__device__
double cuda_noise::GradientNoise3D (double fx, double fy, double fz, int ix,
  int iy, int iz, int seed)
{
  // Randomly generate a gradient vector given the integer coordinates of the
  // input value.  This implementation generates a random number and uses it
  // as an index into a normalized-vector lookup table.
  int vectorIndex = (
      X_NOISE_GEN    * ix
    + Y_NOISE_GEN    * iy
    + Z_NOISE_GEN    * iz
    + SEED_NOISE_GEN * seed)
    & 0xffffffff;
  vectorIndex ^= (vectorIndex >> SHIFT_NOISE_GEN);
  vectorIndex &= 0xff;

  double xvGradient = g_randomVectors[(vectorIndex << 2)    ];
  double yvGradient = g_randomVectors[(vectorIndex << 2) + 1];
  double zvGradient = g_randomVectors[(vectorIndex << 2) + 2];

  // Set up us another vector equal to the distance between the two vectors
  // passed to this function.
  double xvPoint = (fx - (double)ix);
  double yvPoint = (fy - (double)iy);
  double zvPoint = (fz - (double)iz);

  // Now compute the dot product of the gradient vector with the distance
  // vector.  The resulting value is gradient noise.  Apply a scaling value
  // so that this noise value ranges from -1.0 to 1.0.
  return ((xvGradient * xvPoint)
    + (yvGradient * yvPoint)
    + (zvGradient * zvPoint)) * 2.12;
}

__device__
int cuda_noise::IntValueNoise3D (int x, int y, int z, int seed)
{
  // All constants are primes and must remain prime in order for this noise
  // function to work correctly.
  int n = (
      X_NOISE_GEN    * x
    + Y_NOISE_GEN    * y
    + Z_NOISE_GEN    * z
    + SEED_NOISE_GEN * seed)
    & 0x7fffffff;
  n = (n >> 13) ^ n;
  return (n * (n * n * 60493 + 19990303) + 1376312589) & 0x7fffffff;
}

__device__
double cuda_noise::ValueCoherentNoise3D (double x, double y, double z, int seed)
{
  // Create a unit-length cube aligned along an integer boundary.  This cube
  // surrounds the input point.
  int x0 = (x > 0.0? (int)x: (int)x - 1);
  int x1 = x0 + 1;
  int y0 = (y > 0.0? (int)y: (int)y - 1);
  int y1 = y0 + 1;
  int z0 = (z > 0.0? (int)z: (int)z - 1);
  int z1 = z0 + 1;

  // Map the difference between the coordinates of the input value and the
  // coordinates of the cube's outer-lower-left vertex onto an S-curve.
  double xs = 0, ys = 0, zs = 0;

  xs = SCurve3 (x - (double)x0);
  ys = SCurve3 (y - (double)y0);
  zs = SCurve3 (z - (double)z0);

  // Now calculate the noise values at each vertex of the cube.  To generate
  // the coherent-noise value at the input point, interpolate these eight
  // noise values using the S-curve value as the interpolant (trilinear
  // interpolation.)
  double n0, n1, ix0, ix1, iy0, iy1;
  n0   = ValueNoise3D (x0, y0, z0, seed);
  n1   = ValueNoise3D (x1, y0, z0, seed);
  ix0  = LinearInterp (n0, n1, xs);
  n0   = ValueNoise3D (x0, y1, z0, seed);
  n1   = ValueNoise3D (x1, y1, z0, seed);
  ix1  = LinearInterp (n0, n1, xs);
  iy0  = LinearInterp (ix0, ix1, ys);
  n0   = ValueNoise3D (x0, y0, z1, seed);
  n1   = ValueNoise3D (x1, y0, z1, seed);
  ix0  = LinearInterp (n0, n1, xs);
  n0   = ValueNoise3D (x0, y1, z1, seed);
  n1   = ValueNoise3D (x1, y1, z1, seed);
  ix1  = LinearInterp (n0, n1, xs);
  iy1  = LinearInterp (ix0, ix1, ys);
  return LinearInterp (iy0, iy1, zs);
}

__device__
double cuda_noise::ValueNoise3D (int x, int y, int z, int seed)
{
  return 1.0 - ((double)IntValueNoise3D (x, y, z, seed) / 1073741824.0);
}

