#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "mapgen.cuh"
#include "noisegen.cuh"
#include "noise_helpers.cuh"
#include "hip/hip_runtime_api.h"

#define THREADS_PER_BLOCK 128;

#define LAND_SLOPE 20
#define SHORE_SIZE 40

#define AIR 0
#define WATER 1
#define DIRT 2
#define STONE 8
#define GRANITE 9
#define BASALT 10
#define PERIDOTITE 13
#define GLOWSTONE 22

/*Source: 
http://stackoverflow.com/questions/17399119/
cant-we-use-atomic-operations-for-floating-point-variables-in-cuda
*/
/*__device__ static float atomicMax(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}*/

__device__
int getX(int width) {
    return (blockIdx.x * blockDim.x + threadIdx.x) % width;
}

__device__
int getY(int width) {
    return (blockIdx.x * blockDim.x + threadIdx.x) / width;
}

__device__
double surface_modifier(int x, int y, int width, int height) {
    int baseHeight = height * 0.8;
    int seafloorLevel = height * 0.5;
    int shoreline = width * 0.25;
    int abyss = width * 0.35;
    double steepness = 50;
    // Baseline is at baseHeight
    double surface = (y - baseHeight) / steepness;
    // Continental modifier
    double quadratic = LAND_SLOPE * ((x - width / 2.0) 
            * (x - width / 2.0)) / (width * width);
    // Oceanic modifier
    double linear = 5 * abs(width / 2.0 - x) / width;
    // Ocean floor's baseline is at seafloorLevel and it gets a little deeper linearly
    int depth = (baseHeight - seafloorLevel) / steepness;
    if (abs(width / 2.0 - x) > abyss) {
        surface += depth + linear;
    }
    // Interpolate between land height and seafloor depth
    else if (abs(width / 2.0 - x) > shoreline) {
        double dist = (abs(width / 2.0 - x) - shoreline) 
                        / (abyss - shoreline);
        double interp = dist < 0.5? pow(0.5 - dist, 1 / 3.0) 
            : -1 * pow(dist - 0.5, 1 / 3.0);
        interp /= 2 * 0.7937; //cube root of 0.5
        interp += 0.5;
        surface += (1 - interp) * (depth + linear);
        surface += interp * quadratic;
    }
    // This gives a slight curve to the land
    else {
        surface += quadratic;
    }
    return surface;
}

// Ideally would increase the turbulence on here a bit
__device__
double cave_noise(int seed, int xi, int yi, int width, int height) {
    seed += 20;
    double scale = 20.0 / 2048.0;
    double y_scale = 2.0;
    double x, y, z;
    adjustLocation(xi, yi, width, height, scale, &x, &y, &z);
    y *= y_scale;

    return cuda_noise::GetRidgedMultiValue(x, y, z, seed);
}

// This is mapgen.cc's tunnel
__device__
double cavern_noise(int seed, int xi, int yi, int width, int height) {
    seed += 30;
    double scale = 2.7 / 2048.0;
    double y_scale = 3.0;
    double x, y, z;
    adjustLocation(xi, yi, width, height, scale, &x, &y, &z);
    y *= y_scale;

    return cuda_noise::GetRidgedMultiValue(x, y, z, seed);
}

__device__
double surface_noise(int seed, int xi, int yi, int width, int height) {
    seed += 40;
    double scale = 3.0 / 2048.0;
    double x, y, z;
    adjustLocation(xi, yi, width, height, scale, &x, &y, &z);

    return cuda_noise::GetPerlinValue(x, y, z, seed);
}

// This is mapgen.cc's wetness + humidity
__device__
double wetness_noise(int seed, int xi, int yi, int width, int height) {
    int wet_seed = seed + 50; // scaled by 0.015
    int humid_seed = seed - 10; // scaled by 0.0014
    double wet_scale = 15 * 3.0 / 2048;
    double humid_scale = 1.4 * 3.0 / 2048;

    double x, y, z;
    adjustLocation(xi, yi, width, height, humid_scale, &x, &y, &z);
    double value = cuda_noise::GetPerlinValue(x, y, z, humid_seed);
    adjustLocation(xi, yi, width, height, wet_scale, &x, &y, &z);
    return value + cuda_noise::GetPerlinValue(x, y, z, wet_seed);
}

__device__
double felsic_noise(int seed, int xi, int yi, int width, int height) {
    seed += 60;
    double scale = 3.0 / 2048.0;
    
    double x, y, z;
    adjustLocation(xi, yi, width, height, scale, &x, &y, &z);

    return cuda_noise::GetPerlinValue(x, y, z, seed);
}

__device__
bool is_cavern(double surface, double cavern, int y, int height) {
    const double cavernBoundary = 0.66;
    const double cavernLimit = -1.63;
    int cavernHeight = height * 0.5;
    double s = max(surface, (y - cavernHeight) / 50.0 + surface);

    return cavern > cavernBoundary && s / 2.0 - cavern < cavernLimit;
}

__device__
char choose_rock(int seed, int xi, int yi, int width, int height) {
    const double basaltLimit = -0.31;
    const double graniteLimit = 0.31;
    const double peridotLimit = -0.73;
    const int seafloorLevel = height * 0.5;
    const int baseHeight = height * 0.8;
    char rock = STONE;

    double interp = 0;
    double felsic = felsic_noise(seed, xi, yi, width, height);
    // The CPU code relies on the surfaces vector, which could still be 
    // being written, so I'm using a quick approximation of surface_modifier instead
    int shoreline = width * 0.25;
    int abyss = width * 0.35;
    int surface;
    if (abs(xi - (width / 2.0)) < shoreline) {
        surface = baseHeight;
    }
    else if (abs(xi - (width / 2.0)) > abyss) {
        surface = seafloorLevel;
    }
    else {
        interp = (abs(xi - (width / 2.0)) - shoreline) / (abyss - shoreline);
        surface = interp * seafloorLevel + (1 - interp) * baseHeight;
    }

    // Now mirror it
    /* Adjust so that continental plates tend to be made of 
    granite, while oceanic plates tend to be made of basalt, 
    and the upper mantle is peridotite. */
    if (seafloorLevel - yi > surface - seafloorLevel) {
        double dist = surface > seafloorLevel? 
            2 * seafloorLevel - surface : surface;
        interp = abs((dist - yi) / dist);
        felsic -= abs(peridotLimit + basaltLimit) / 2.0 + interp;
    }
    else if (seafloorLevel - yi == surface - seafloorLevel) {
        // pass
    }
    else {
        double dist = 2 * (surface - seafloorLevel);
        interp = abs((dist - (surface - yi)) / dist);
        felsic += abs(graniteLimit) / 2.0 + 0.2 * interp;
    }

    if (felsic < peridotLimit && interp - 0.7 > 0.25 * felsic) {
        rock = PERIDOTITE;
    }
    else if (felsic < basaltLimit) {
        rock = BASALT;
    }
    else if (felsic > graniteLimit) {
        rock = GRANITE;
    }
    return rock;
}

__global__
void dev_generate_terrain(int seed, int width, int height, char *map, int *surfaces) {
    const double caveBoundary = 0.55;
    const double caveLimit = -1.24;
    const double waterLimit = 0.83;

    const int i = getX(width);
    const int j = getY(width);
    
    char tile = STONE;
    double surface = surface_modifier(i, j, width, height) 
                    + surface_noise(seed, i, j, width, height);
    double cave = cave_noise(seed, i, j, width, height);
    double cavern = cavern_noise(seed, i, j, width, height);
    if (surface > 0
            || cave > caveBoundary && surface - cave < caveLimit) {
        tile = AIR;
    }
    else if (is_cavern(surface, cavern, j, height)) {
        if (j + 1 < height) {
            double up_surface = surface_modifier(i, j + 1, width, height) 
                        + surface_noise(seed, i, j + 1, width, height);
            double up_cavern = cavern_noise(seed, i, j + 1, width, height);
            if (!is_cavern(up_surface, up_cavern, j + 1, height)) {
                tile = GLOWSTONE;
            }
            else {
                tile = AIR;
            }
        }
    }
    else {
        atomicMax(&surfaces[i], j);
        tile = choose_rock(seed, i, j, width, height);
    }

    if (tile == AIR && wetness_noise(seed, i, j, width, height) > waterLimit) {
        tile = WATER;
    }

    map[blockIdx.x * blockDim.x + threadIdx.x] = tile;
}

char *generate_terrain(int seed, int width, int height, int *surfaces) {
    char *dev_map;
    int *dev_surfaces;
    CUDA_CALL(hipMalloc(&dev_map, width * height * sizeof(char)));
    CUDA_CALL(hipMalloc(&dev_surfaces, width * sizeof(int)));

    int n_blocks = width * height / THREADS_PER_BLOCK;
    dim3 num_blocks(n_blocks);
    dim3 num_threads(128);

    //int seaLevel = height * 0.72;
 
    dev_generate_terrain<<<num_blocks, num_threads>>>(seed, width, height, dev_map, dev_surfaces);

    char *map = (char *)malloc(width * height * sizeof(char));
    CUDA_CALL(hipMemcpy(map, dev_map, width * height * sizeof(char), hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(surfaces, dev_surfaces, width * sizeof(int), hipMemcpyDeviceToHost));
    return map;
}

