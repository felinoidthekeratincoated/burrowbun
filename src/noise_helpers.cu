#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "noise_helpers.cuh"
#include "noisegen.cuh"
#include <math.h>


__device__
void adjustLocation(int xi, int yi, int width, int height, double scale, double *x_out, double *y_out, double *z_out) {
    // Wrap the map around a cylinder so the edges match
    double angle = xi * 2.0 * M_PI / width;

    // Transform to a cylinder of diameter width and height height
    // This is different from libnoise's cylinder, which is the unit, but I
    // think this is a better way of doing it (even though I have to 
    // readjust all the scales).
    double factor = width / (2.0 * M_PI);
    double x = cos(angle) * factor;
    double y = yi;
    double z = sin(angle) * factor;

    // Every single call includes turbulence and scale, so might as well do that here

    // Turbulence function modified from libnoise's turbulence module

    // Default value from libnoise
    int seed = 0;
    double power = 1.0;

    double x0, y0, z0;
    double x1, y1, z1;
    double x2, y2, z2;
    x0 = x + (12414.0 / 65536.0);
    y0 = y + (65124.0 / 65536.0);
    z0 = z + (31337.0 / 65536.0);
    x1 = x + (26519.0 / 65536.0);
    y1 = y + (18128.0 / 65536.0);
    z1 = z + (60493.0 / 65536.0);
    x2 = x + (53820.0 / 65536.0);
    y2 = y + (11213.0 / 65536.0);
    z2 = z + (44845.0 / 65536.0);
    double xDistort = x + (cuda_noise::GetPerlinValue(x0, y0, z0, seed)
    * power * factor);
    double yDistort = y + (cuda_noise::GetPerlinValue(x1, y1, z1, seed + 1)
    * power);
    double zDistort = z + (cuda_noise::GetPerlinValue(x2, y2, z2, seed + 2)
    * power * factor);
    
    // scale
    x *= scale;
    y *= scale;
    z *= scale;

    // And set outputs
    *x_out = x;
    *y_out = y;
    *z_out = z;
}
